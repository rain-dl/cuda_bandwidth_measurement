﻿#include <iostream>

#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"


namespace kernel
{

	__global__ void measure_global_bandwidth_kb(int *out, int *device, int size)
	{
		int r = 0;
		for (int i = 0; i < size; ++i)
		{
			r += device[i];
		}
		*out = r;
	}

}

auto measure_host_device_bandwidth_mb(const int n, const int repeat, const bool enable_sync)
{
	const int bytes = n * (1 << 20);

	int *host, *device;
	hipHostMalloc((void **)&host, bytes, hipHostMallocDefault);
	hipMalloc((void **)&device, bytes);

	hipEvent_t hd_start, hd_stop, dh_start, dh_stop;
	hipEventCreate(&hd_start);
	hipEventCreate(&hd_stop);
	hipEventCreate(&dh_start);
	hipEventCreate(&dh_stop);

	float hd_sum = 0, dh_sum = 0;

	for (int i = 0; i < repeat; i++)
	{
		float hd_time, dh_time;

		if (enable_sync)
		{
			hipEventRecord(hd_start);
			hipMemcpy(device, host, bytes, hipMemcpyHostToDevice);
			hipEventRecord(hd_stop);
			hipEventSynchronize(hd_stop);

			hipEventRecord(dh_start);
			hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost);
			hipEventRecord(dh_stop);
			hipEventSynchronize(dh_stop);
		}
		else
		{
			hipEventRecord(hd_start);
			hipMemcpyAsync(device, host, bytes, hipMemcpyHostToDevice);
			hipEventRecord(hd_stop);
			hipEventSynchronize(hd_stop);

			hipEventRecord(dh_start);
			hipMemcpyAsync(host, device, bytes, hipMemcpyDeviceToHost);
			hipEventRecord(dh_stop);
			hipEventSynchronize(dh_stop);
		}

		hipEventElapsedTime(&hd_time, hd_start, hd_stop);
		hipEventElapsedTime(&dh_time, dh_start, dh_stop);

		hd_sum += hd_time;
		dh_sum += dh_time;
	}

	hipHostFree(host);
	hipFree(device);

	hipEventDestroy(hd_start);
	hipEventDestroy(hd_stop);
	hipEventDestroy(dh_start);
	hipEventDestroy(dh_stop);

	return std::make_pair(hd_sum, dh_sum);
}

void measure_host_device_bandwidth(const bool enable_sync)
{
	const int repeat = 100;
	std::cout << "host <-> device  " << (enable_sync ? "sync" : "aysnc") << std::endl;
	std::cout << "data size[MB], host to device[ms], host to device[GB/s], device to host[ms], device to host[GB/s]" << std::endl;
	for (int n = 32; n <= 256; n += 32)
	{
		const auto time = measure_host_device_bandwidth_mb(n, repeat, enable_sync);
		float hd_sum = time.first;
		float dh_sum = time.second;
		std::cout << n << ", " << hd_sum / repeat << ", " << n * 1000 * repeat / hd_sum / 1024 << ", " << dh_sum / repeat << ", " << n * 1000 * repeat / dh_sum / 1024 << std::endl;
	}
	std::cout << "--\n"
			  << std::endl;
}

auto measure_global_bandwidth_kb(const int n)
{
	const int bytes = n * (1 << 20);

	int *out, *device;
	hipMalloc((void **)&out, sizeof(int));
	hipMalloc((void **)&device, bytes);

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 grid(1);
	dim3 threads(1);

	hipEventRecord(start);
	kernel::measure_global_bandwidth_kb<<<grid, threads>>>(out, device, bytes / sizeof(int));
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);

	hipFree(out);
	hipFree(device);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time;
}

void measure_global_bandwidth()
{
	const int repeat = 100;
	std::cout << "global memory" << std::endl;
	std::cout << "data size[MB], time[ms], bandwidth[GB/s]" << std::endl;
	for (int n = 32; n <= 512; n += 32)
	{
		float sum = 0;
		for (int i = 0; i < repeat; ++i)
		{
			sum += measure_global_bandwidth_kb(n);
		}

		std::cout << n << ", " << sum / repeat << ", " << n * 1000 * repeat / sum / 1024 << std::endl;
	}
	std::cout << "--\n"
			  << std::endl;
}

int main()
{
	measure_host_device_bandwidth(true);
	measure_host_device_bandwidth(false);
	measure_global_bandwidth();
	return 0;
}
